#include "hip/hip_runtime.h"
/* GStreamer
 * Copyright (C) 2025 Seungha Yang <seungha@centricular.com>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Library General Public
 * License as published by the Free Software Foundation; either
 * version 2 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Library General Public License for more details.
 *
 * You should have received a copy of the GNU Library General Public
 * License along with this library; if not, write to the
 * Free Software Foundation, Inc., 51 Franklin St, Fifth Floor,
 * Boston, MA 02110-1301, USA.
 */

#ifdef __NVCC__
__device__ inline unsigned char
scale_to_uchar (float val)
{
  return (unsigned char) __float2int_rz (val * 255.0);
}

extern "C" {
__global__ void
GstNvJpegEncConvertMain (hipTextureObject_t uv_tex, unsigned char * out_u,
    unsigned char * out_v, int width, int height, int stride)
{
  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_pos >= width || y_pos >= height)
    return;
  float x = 0;
  float y = 0;
  if (width > 1)
    x = (float) x_pos / (width - 1);
  if (height > 1)
    y = (float) y_pos / (height - 1);
  float2 uv = tex2D<float2> (uv_tex, x, y);
  unsigned int pos = x_pos + (y_pos * stride);
  out_u[pos] = scale_to_uchar (uv.x);
  out_v[pos] = scale_to_uchar (uv.y);
}
}
#else
static const gchar *GstNvJpegEncConvertMain_str = R"(
__device__ inline unsigned char
scale_to_uchar (float val)
{
  return (unsigned char) __float2int_rz (val * 255.0);
}

extern "C" {
__global__ void
GstNvJpegEncConvertMain (hipTextureObject_t uv_tex, unsigned char * out_u,
    unsigned char * out_v, int width, int height, int stride)
{
  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_pos >= width || y_pos >= height)
    return;
  float x = 0;
  float y = 0;
  if (width > 1)
    x = (float) x_pos / (width - 1);
  if (height > 1)
    y = (float) y_pos / (height - 1);
  float2 uv = tex2D<float2> (uv_tex, x, y);
  unsigned int pos = x_pos + (y_pos * stride);
  out_u[pos] = scale_to_uchar (uv.x);
  out_v[pos] = scale_to_uchar (uv.y);
}
}
)";
#endif
